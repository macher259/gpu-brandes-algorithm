#include "brandes.cuh"

brandes::AlgorithmContext::AlgorithmContext(unsigned int real_count, unsigned int virtual_count_, unsigned int edge_count_) {
    offset = new unsigned int[virtual_count_];
    virtual_map = new unsigned int[virtual_count_];
    virtual_count = new unsigned int[real_count];
    pointers = new unsigned int[real_count + 1];
    adjacent = new unsigned int[edge_count_];
    bc_values = new double[real_count];
    std::fill(bc_values, bc_values + real_count, 0.0);
    reach = new unsigned int[real_count];
    std::fill(reach, reach + real_count, 1);
}

brandes::AlgorithmContext::~AlgorithmContext() {
    delete offset;
    delete virtual_map;
    delete virtual_count;
    delete pointers;
    delete adjacent;
    delete bc_values;
    delete reach;
}

brandes::DeviceAlgorithmContext::DeviceAlgorithmContext(brandes::AlgorithmContext &host_context, unsigned int real_count_, unsigned int virtual_count_, unsigned int edge_count_)
: distance_size(real_count_ * sizeof(unsigned int)), sigma_size(real_count_ * sizeof(unsigned int)), delta_size(real_count_ * sizeof(double))
{
    auto offset_size = virtual_count_ * sizeof(unsigned int);
    HANDLE_ERROR(hipMalloc(&offset, offset_size));
    HANDLE_ERROR(hipMemcpy(offset, host_context.offset, offset_size, hipMemcpyHostToDevice));

    auto vmap_size = virtual_count_ * sizeof(unsigned int);
    HANDLE_ERROR(hipMalloc(&virtual_map, vmap_size));
    HANDLE_ERROR(hipMemcpy(virtual_map, host_context.virtual_map, vmap_size, hipMemcpyHostToDevice));

    auto nvir_size = real_count_ * sizeof(unsigned int);
    HANDLE_ERROR(hipMalloc(&virtual_count, nvir_size));
    HANDLE_ERROR(hipMemcpy(virtual_count, host_context.virtual_count, nvir_size, hipMemcpyHostToDevice));

    auto ptrs_size = (real_count_ + 1) * sizeof(unsigned int);
    HANDLE_ERROR(hipMalloc(&pointers, ptrs_size));
    HANDLE_ERROR(hipMemcpy(pointers, host_context.pointers, ptrs_size, hipMemcpyHostToDevice));

    auto adjs_size = edge_count_ * sizeof(unsigned int);
    HANDLE_ERROR(hipMalloc(&adjacent, adjs_size));
    HANDLE_ERROR(hipMemcpy(adjacent, host_context.adjacent, adjs_size, hipMemcpyHostToDevice));

    auto bc_size = real_count_ * sizeof(double);
    HANDLE_ERROR(hipMalloc(&bc_values, bc_size));
    HANDLE_ERROR(hipMemset(bc_values, 0.0, bc_size));

    HANDLE_ERROR(hipMalloc(&distance, distance_size));
    HANDLE_ERROR(hipMalloc(&sigma, sigma_size));
    HANDLE_ERROR(hipMalloc(&delta, delta_size));
    HANDLE_ERROR(hipMalloc(&should_continue, sizeof(bool)));

    HANDLE_ERROR(hipMalloc(&reach, sizeof(unsigned int) * real_count_));
    HANDLE_ERROR(hipMemcpy(reach, host_context.reach, sizeof(unsigned int) * real_count_, hipMemcpyHostToDevice));
}

void brandes::DeviceAlgorithmContext::initializeForSource(unsigned int source) const {
    HANDLE_ERROR(hipMemset(distance, NOT_VISITED, distance_size));
    HANDLE_ERROR(hipMemset(sigma, 0, sigma_size));
    HANDLE_ERROR(hipMemset(&distance[source], 0, sizeof(unsigned int)));
    HANDLE_ERROR(hipMemset(&sigma[source], 1, 1));
}

brandes::DeviceAlgorithmContext::~DeviceAlgorithmContext() {
    HANDLE_ERROR(hipFree(offset));
    HANDLE_ERROR(hipFree(virtual_map));
    HANDLE_ERROR(hipFree(virtual_count));
    HANDLE_ERROR(hipFree(pointers));
    HANDLE_ERROR(hipFree(adjacent));
    HANDLE_ERROR(hipFree(bc_values));
    HANDLE_ERROR(hipFree(distance));
    HANDLE_ERROR(hipFree(sigma));
    HANDLE_ERROR(hipFree(delta));
    HANDLE_ERROR(hipFree(should_continue));
    HANDLE_ERROR(hipFree(reach));
}
